#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdbool.h>
#include "mmio.h"
#include "mmiohighlevel.h"

#include "hipblas.h"
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 16

typedef struct
{
	VALUE_TYPE *value;
	int *columnindex;
	int *rowpointer;

} SMatrix;

__global__ void BiasReLU(int row, int col, VALUE_TYPE *data_C, VALUE_TYPE *data_A, VALUE_TYPE bias)
{
	int tmpRow = blockDim.x * blockIdx.x + threadIdx.x;
	int tmpCol = blockDim.y * blockIdx.y + threadIdx.y;
	int tmpPos = tmpRow * col + tmpCol;

	if (tmpPos < row * col)
	{
		data_A[tmpPos] = data_C[tmpPos] + bias;

		if (data_A[tmpPos] <= 0) {
			data_A[tmpPos] = 0;
		} else if (data_A[tmpPos] >= 32) {
			data_A[tmpPos] = 32;
		}
	}
}

int main(int argc, char **argv)
{
	struct timeval t1, t2, t3, t4;
	int size1 = 0;
	int size2 = 0;
	int *tc1;
	int *tc2;
	double bias = -0.3000;

	int mA;
	int nA;
	int nnzA;
	int isSymmetricA;
	SMatrix A;

	int mB;
	int nB;
	int nnzB;
	int isSymmetricB;
	SMatrix B[120];

	int mC, nC;
	int nnzC_golden = 0;

	// load matrix data from file
	gettimeofday(&t3, NULL);
	char filename1[] = "sparse-images-1024.tsv";
	mmio_info(&mA, &nA, &nnzA, &isSymmetricA, filename1);
	A.value = (VALUE_TYPE *)malloc((nnzA) * sizeof(VALUE_TYPE));
	A.columnindex = (int *)malloc((nnzA) * sizeof(int));
	A.rowpointer = (int *)malloc((mA + 1) * sizeof(int));
	mmio_data(A.rowpointer, A.columnindex, A.value, filename1);
	printf("input matrix A: ( %i, %i ) nnz = %i\n", mA, nA, nnzA);
	VALUE_TYPE *A0 = (VALUE_TYPE *)malloc(mA * nA * sizeof(VALUE_TYPE));
	memset(A0, 0, sizeof(VALUE_TYPE) * mA * nA);
	for (int i = 0; i < mA; i++)
	{
		for (int j = A.rowpointer[i]; j < A.rowpointer[i + 1]; j++)
		{
			A0[i * nA + A.columnindex[j]] = A.value[j];
		}
	}
	free(A.rowpointer);
	free(A.columnindex);
	free(A.value);

	char neuronfile1[] = "neuron1024/n1024-l";
	char neuronfile2[] = ".tsv";
	char filename3[60];

	VALUE_TYPE *B0[120];
	for (int k = 0; k < 120; k++)
	{
		char filenum[5];
		int k1 = k + 1;
		snprintf(filenum, sizeof(filenum), "%d", k1);

		strcpy(filename3, neuronfile1);
		strcat(filename3, filenum);
		strcat(filename3, neuronfile2);

		mmio_info(&mB, &nB, &nnzB, &isSymmetricB, filename3);
		B[k].value = (VALUE_TYPE *)malloc((nnzB) * sizeof(VALUE_TYPE));
		B[k].columnindex = (int *)malloc((nnzB) * sizeof(int));
		B[k].rowpointer = (int *)malloc((mB + 1) * sizeof(int));
		mmio_data(B[k].rowpointer, B[k].columnindex, B[k].value, filename3);

		B0[k] = (VALUE_TYPE *)malloc(mB * nB * sizeof(VALUE_TYPE));
		memset(B0[k], 0, sizeof(VALUE_TYPE) * mB * nB);
		for (int i = 0; i < mB; i++)
		{
			for (int j = B[k].rowpointer[i]; j < B[k].rowpointer[i + 1]; j++)
			{
				B0[k][i * nB + B[k].columnindex[j]] = B[k].value[j];
			}
		}

		free(B[k].rowpointer);
		free(B[k].columnindex);
		free(B[k].value);
	}
	gettimeofday(&t4, NULL);
	double time_load = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Weight matrix load time: %f ms \n", time_load);

	printf("input matrix B: ( %i, %i ) nnz = %i\n", mB, nB, nnzB);

	mC = mA;
	nC = nB;
	VALUE_TYPE *C0 = (VALUE_TYPE *)malloc((mC * nC) * sizeof(VALUE_TYPE));

	gettimeofday(&t3, NULL);

	// 在GPU中申请空间，并将值复制进
	VALUE_TYPE *d_A, *d_B[120], *d_C;
	hipMalloc((void **)&d_A, sizeof(VALUE_TYPE) * mA * nA);
	hipMalloc((void **)&d_C, sizeof(VALUE_TYPE) * mC * nC);
	hipMemcpy(d_A, A0, sizeof(VALUE_TYPE) * mA * nA, hipMemcpyHostToDevice);
	for (int k = 0; k < 120; k++)
	{
		size_t size = nA * nB * sizeof(VALUE_TYPE);
		hipMalloc((void **)&d_B[k], size);
		hipMemcpy(d_B[k], B0[k], size, hipMemcpyHostToDevice);
	}

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	VALUE_TYPE alpha = 1.0, beta = 0.0;

	dim3 block{BLOCK_SIZE, BLOCK_SIZE};
	dim3 grid{(uint)ceil((VALUE_TYPE)mC / block.x), (uint)ceil((VALUE_TYPE)nC / block.y)};

	// gettimeofday(&t3, NULL);

// 计算 GEMM
	for (int k = 0; k < 120; k++)
	{
		int k1 = k + 1;

		gettimeofday(&t1, NULL);
		hipDeviceSynchronize();
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nB, mA, mB, &alpha, d_B[k], nB, d_A, nA, &beta, d_C, nB);
		hipDeviceSynchronize();
		gettimeofday(&t2, NULL);
		double time_gemm = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;

		gettimeofday(&t1, NULL);
		hipDeviceSynchronize();
		BiasReLU<<<grid, block>>>(mC, nC, d_C, d_A, bias);
		hipDeviceSynchronize();
		gettimeofday(&t2, NULL);
		double time_biasrelu = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
		printf("k = %d, GEMM time: %4.5f ms, Bias+ReLU time: %4.5f ms\n",
			   k + 1, time_gemm, time_biasrelu);
	}

	hipMemcpy(A0, d_A, sizeof(VALUE_TYPE) * mC * nC, hipMemcpyDeviceToHost);
	gettimeofday(&t4, NULL);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	double time_inference = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Inference time: %f ms \n", time_inference);

	free(C0);

	// check results
	printf("test\n");
	FILE *fs;
	fs = fopen("sparse-images-1024-1.tsv", "w+");
	for (int i = 0; i < mA; i++)
	{
		int sum = 0;
		for (int j = (i * nA); j < ((i + 1) * nA); j++)
		{
			sum += A0[j];
		}
		if (sum != 0)
		{
			fprintf(fs, "%d\n", i + 1);
		}
	}
	fclose(fs);
	FILE *fp2 = NULL;

	fp2 = fopen("sparse-images-1024-1.tsv", "rb");
	if (fp2 == NULL)
	{
		printf("Error:Open file fail!\n");
	}

	fseek(fp2, 0, SEEK_END);
	size2 = ftell(fp2);
	rewind(fp2);

	tc2 = (int *)malloc(sizeof(int) * size2 / 4);

	int readnum2 = fread(tc2, 4, size2 / 4, fp2);

	fclose(fp2);

	FILE *fp1;

	fp1 = fopen("neuron1024-l120-categories.tsv", "rb");
	if (fp1 == NULL)
	{
		printf("Error:Open file fail!\n");
	}

	fseek(fp1, 0, SEEK_END);
	size1 = ftell(fp1);
	rewind(fp1);

	tc1 = (int *)malloc(sizeof(int) * size1 / 4);

	int readnum1 = fread(tc1, 4, size1 / 4, fp1);

	fclose(fp1);
	int judge = 0;
	for (int i = 0; i < size1 / 4; i++)
	{
		if (tc1[i] - tc2[i] != 0)
		{
			judge++;
		}
	}
	printf("judge:%d\n", judge);
	if (judge == 0)
	{
		printf("CHALLENGE PASSED\n");
	}
	else
	{
		printf("CHALLENGE FAILED\n");
	}

	free(A0);

	return 0;
}
